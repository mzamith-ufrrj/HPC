#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
#include <math.h>
#include <string.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <App.h>
#include <hip/hip_runtime_api.h>
#define BLOCK_SIZE 1024
//#define BLOCK_SIZE 1024
#define EPSILON    1E-9
#define G          1.0
#define DT         1.5
/*Declarando as structs de particula e forca*/
struct stCoord{
    float x,
          y,
          z;
};
typedef struct stCoord tpCoord;
struct stParticle
{
    tpCoord p,
            v,
            f;
    float m;
};
typedef struct stParticle tpParticle;


void printLog(tpParticle *particles, int nParticles){
    char fileName[128];
    sprintf(fileName, "%s.txt", __FILE__);
    fprintf(stdout, "Saving file [%s] ", fileName); fflush(stdout);
    FILE *ptr = fopen(fileName, "w+");
      for(int i = 0; i < nParticles; i++){
            fprintf(ptr, "%d \t %.10f %.10f %.10f \t %.10f %.10f %.10f \t %.10f %.10f %.10f \n", i,  particles[i].p.x, particles[i].p.y, particles[i].p.z,  particles[i].v.x, particles[i].v.y, particles[i].v.z, particles[i].f.x, particles[i].f.y, particles[i].f.z);

    }
    fclose(ptr);
    fprintf(stdout, "[OK]\n"); fflush(stdout);
}


void initialCondition(tpParticle *particles, int nParticles){

    srand(42);

    memset(particles, 0x00, nParticles * sizeof(tpParticle));
    
    for (int i = 0; i < nParticles ; i++){
        particles[i].p.x =  2.0 * (rand() / (float)RAND_MAX) - 1.0;
        particles[i].p.y =  2.0 * (rand() / (float)RAND_MAX) - 1.0;
        particles[i].p.z =  2.0 * (rand() / (float)RAND_MAX) - 1.0;
        particles[i].m =  1;
     
     
      }




}

__device__ float distance(  float* dx,
                            float* dy,
                            float* dz,
                            const tpParticle A,
                            const tpParticle B){
    float x = A.p.x - B.p.x;
    float y = A.p.y - B.p.y;
    float z = A.p.z - B.p.z;
    *dx = x;
    *dy = y;
    *dz = z;
    float d = (x * x) + (y * y) + (z * z) + EPSILON;
    return d;

}

__global__ void particleParticle_k(tpParticle *particles){
    extern __shared__ tpParticle subParticles[];
    int i  = blockDim.x * blockIdx.x + threadIdx.x;
//  __shared__ tpParticle subParticles[BLOCK_SIZE];
    float sum_fx = 0.0,
          sum_fy = 0.0,
          sum_fz = 0.0,
          mi = particles[i].m;

    for (int blk = 0; blk < gridDim.x; blk++){
        subParticles[threadIdx.x] = particles[ blockDim.x * blk + threadIdx.x];
        __syncthreads();

        for (int j = 0; j < blockDim.x; j++){
            float dx = 0.0f,  dy = 0.0f, dz = 0.0f;
            float d  = distance(&dx, &dy, &dz, particles[i], subParticles[j]);
            float mj = subParticles[j].m;
            sum_fx += (dx * mj * mi) / (d * d * d) ;
            sum_fy += (dy * mj * mi) / (d * d * d) ;
            sum_fz += (dz * mj * mi) / (d * d * d) ;
        }//end-for (int j = 0; j < blockDim.x; j++){

        __syncthreads();
        particles[i].f.x = -G * sum_fx;
        particles[i].f.y = -G * sum_fy;
        particles[i].f.z =- G * sum_fz;
    }//end-for (int blk = 0; blk < gridDim.x; blk++){



    particles[i].v.x += DT * particles[i].f.x;
    particles[i].v.y += DT * particles[i].f.y;
    particles[i].v.z += DT * particles[i].f.z;

    particles[i].p.x += DT * particles[i].v.x;
    particles[i].p.y += DT * particles[i].v.y;
    particles[i].p.z += DT * particles[i].v.z;
}

void saveLog(char *logFile, 
             unsigned long mem, 
             double processor, double total){

   FILE *ptr = NULL;
   struct stat buffer;   
   int ret = stat(logFile, &buffer); 
   if (ret == -1){
      ptr = fopen(logFile, "w+");
      assert(ptr != NULL);
      fprintf(ptr, "memoryUsed;totalTime;processorTime;overheadTime\n");

   }else{
      ptr = fopen(logFile, "a+");
   }
   double overhead = total - processor;
   
   fprintf(ptr, "%lu;%lf;%lf;%lf\n", mem, total, processor, overhead);
   fclose(ptr);
   printf("Arquivo: %s salvo\n", logFile);
}
int main (int ac, char **av){
    int timesteps  = atoi(av[1]),
        nParticles = atoi(av[2]),
        flagSave = atoi(av[4]),
        threads = 0,
        blocks  = 0;

    char logFile[1024];
    Stopwatch   totalTime, processorTime;
    tpParticle *h_particles = NULL,
               *d_particles = NULL;

    START_STOPWATCH(totalTime);
    unsigned long mem = nParticles * sizeof(tpParticle);

    strcpy(logFile, av[3]);

    fprintf(stdout, "\nP2P particle system - CUDA version\n");
    fprintf(stdout, "Memory used %lu bytes - CUDA memory used: %lu \n", mem, mem);
    fprintf(stdout, "Log %s \n", logFile);


    h_particles = (tpParticle *) aligned_alloc(ALING, mem);
    assert(h_particles != NULL);
    initialCondition(h_particles, nParticles);
    
    threads = BLOCK_SIZE,
    blocks  = nParticles / BLOCK_SIZE;
    if (nParticles < 1000){
      blocks = 1;
      threads = nParticles;
    }
    assert(hipDeviceReset()== hipSuccess);
    assert(hipMalloc((void**) &d_particles, mem) == hipSuccess);
    assert(hipMemcpy(d_particles, h_particles, mem,  hipMemcpyHostToDevice) == hipSuccess);
    //assert((nParticles % threads) == 0);
    fprintf(stdout, "\n B(%d) T(%d) \n", blocks, threads);

    START_STOPWATCH(processorTime);
    hipProfilerStart();
    for (int t = 0; t < timesteps; t++){
      particleParticle_k<<<blocks, threads, threads * sizeof(tpParticle)>>>(d_particles);
      assert( hipDeviceSynchronize() == hipSuccess);
    }//end-for (int t = 0; t < timesteps; t++){
    hipProfilerStop();
    STOP_STOPWATCH(processorTime);
    assert(hipMemcpy(h_particles, d_particles, nParticles * sizeof(tpParticle),  hipMemcpyDeviceToHost) == hipSuccess);
    hipFree(d_particles);

    if(flagSave)
        printLog(h_particles, nParticles);
    free(h_particles);

    STOP_STOPWATCH(totalTime);
    saveLog(logFile, mem, processorTime.mElapsedTime, totalTime.mElapsedTime);
    return EXIT_SUCCESS;
}
